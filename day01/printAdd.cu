#include <iostream>
#include <hip/hip_runtime.h>

__global__ void vectorAdd(const float* A, const float* B, float* C, int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        C[idx] = A[idx] + B[idx];
    }
}

int main() {
    const int N = 1024; // Number of elements
    const int size = N * sizeof(float); // Corrected to sizeof(float)

    float* h_A = new float[N];
    float* h_B = new float[N];
    float* h_C = new float[N];

    for (int i = 0; i < N; i++) {
        h_A[i] = 1;
        h_B[i] = i;
    }

    float *d_A, *d_B, *d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // Create CUDA events for timing
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    float milliseconds = 0.0f;

    // Time Host-to-Device (H2D) transfers
    hipEventRecord(start);
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "H2D Time: " << milliseconds << " ms\n";

    // Kernel configuration
    int threadsPerBlock = 256;
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Time kernel execution
    hipEventRecord(start);
    vectorAdd<<<blocksPerGrid, threadsPerBlock>>>(d_A, d_B, d_C, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "Kernel Time: " << milliseconds << " ms\n";

    // Time Device-to-Host (D2H) transfer
    hipEventRecord(start);
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    std::cout << "D2H Time: " << milliseconds << " ms\n";

    // Output results (optional)
    for (int i = N - 10; i < N; i++) {
        std::cout << "C[" << i << "] = " << h_C[i] << std::endl;
    }

    // Cleanup
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    return 0;
}